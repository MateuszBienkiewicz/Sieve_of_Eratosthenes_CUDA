#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <algorithm>
#include <chrono>
using namespace std::chrono;
using namespace std;

__global__
void fun(int n, double sqrt_n, bool* tab, int* result) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    int k;
        for (int i = index; i <= sqrt_n; i += stride)
            //if (tab[i])
            if (i>1)
                for (int j = 2; (k = j * i) <= n; j++)
                    tab[k] = false;
        //for (int i = index; i <= n; i += stride)
            //if (tab[i]) {
                //result[0]++;
                //cout << i << ", ";
            //}
    //return wynik;
}

__global__
void count(int n, bool* tab, int* result) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i <= n; i += stride)
        if (tab[i]) {
            result[index]++;
            //cout << i << ", ";
        }
}

__global__ void sum(int* input)
{
    const int tid = threadIdx.x;

    auto step_size = 1;
    int number_of_threads = blockDim.x;

    while (number_of_threads > 0)
    {
        if (tid < number_of_threads)
        {
            const auto fst = tid * step_size * 2;
            const auto snd = fst + step_size;
            input[fst] += input[snd];
        }

        step_size <<= 1;
        number_of_threads >>= 1;
        __syncthreads();
    }
}

int main() {
    const int gpu_indices_for_count = 128;
    int n;
    int* result;
    bool* tab;
    cout << "Podaj n: ";
    cin >> n;
    double sqrt_n = sqrt(n);
    hipMallocManaged(&result, gpu_indices_for_count * sizeof(int));
    hipMallocManaged(&tab, (n+1) * sizeof(bool));
    fill_n(result, gpu_indices_for_count, 0);
    fill_n(tab, n+1, true);
    auto start = high_resolution_clock::now();
    fun<<<1, gpu_indices_for_count >>>(n, sqrt_n, tab, result);
    hipDeviceSynchronize();
    count<<<1, gpu_indices_for_count >>>(n, tab, result);
    hipDeviceSynchronize();
    sum << <1, gpu_indices_for_count /2 >> > ( result);
    hipDeviceSynchronize();
    result[0] -= 2;
    cout << endl <<"Liczby pierwsze:" << endl;
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    for (int i = 2; i <= n; i++)
        if (tab[i]) {
            cout << i << ", ";
        }
    cout << endl << "Czas: " << duration.count() << endl;
    cout << "Liczb pierwszych w zakresie [2, n] jest: " << result[0];
    hipFree(result);
    hipFree(tab);
    return 0;
}
