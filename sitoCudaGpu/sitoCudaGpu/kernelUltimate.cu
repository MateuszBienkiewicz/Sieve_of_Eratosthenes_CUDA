#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <chrono>
using namespace std::chrono;
using namespace std;

__global__
void fun2(int n, int i, bool* tab) {
    int index; index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride; stride = blockDim.x * gridDim.x;
    int k, j;
    if (index >= 2) j = index; else j = index + stride;
    for (; (k = j * i) <= n; j += stride)
            tab[k] = false;
}

__global__
void fun(int n, double sqrt_n, bool* tab) {
    int index; index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride; stride = blockDim.x * gridDim.x;
    int i;
    const int num_streams = 4;
    hipStream_t st;
    hipStreamCreateWithFlags(&st, hipStreamNonBlocking);
    if (index >= 2) i = index; else i = index + stride;
    for (; i <= sqrt_n; i += stride) {
            fun2 << <40000 / (2 * i) + 1, 1024, 0, st >> > (n, i, tab);
    }
    hipStreamDestroy(st);

}

__global__ void sum(int n, bool* tab, int* result)
{
    int index; index = threadIdx.x;
    int stride; stride = blockDim.x;
    int i;
    __shared__ int wyniki[2048];
    for (int k = index; k < 2048; k += stride) wyniki[k] = 0;
    __syncthreads();
    for (i = index; i <= n; i += stride)
        if (tab[i]) {
            wyniki[index]++;
        }

    __syncthreads();

    int tid; tid = threadIdx.x;

    int step_size; step_size = 1;
    int number_of_threads; number_of_threads = blockDim.x;

    //__shared__ int fst;
    //__shared__ int snd;

    while (number_of_threads > 0)
    {
        if (tid < number_of_threads)
        {
            const auto fst = tid * step_size * 2;
            const auto snd = fst + step_size;
            wyniki[fst] += wyniki[snd];
        }

        step_size <<= 1;
        number_of_threads >>= 1;
        __syncthreads();
    }
    if (tid == 0) {
        wyniki[0] -= 2;
        result[0] = wyniki[0];
    }
}

bool InitCUDA(void)
{
    int count = 0;
    int i = 0;

    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                printf("Device %d supports CUDA %d.%d\n", i, prop.major, prop.minor);
                printf("It has warp size %d, %d regs per block, %d bytes of shared memory per block, %d threads per block.\n",
                    prop.warpSize, prop.regsPerBlock, prop.sharedMemPerBlock, prop.maxThreadsPerBlock);
                printf("max Threads %d x %d x %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
                printf("max Grid %d x %d x %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
                printf("total constant memory %d\n", prop.totalConstMem);
                if (prop.concurrentKernels == 0)
                {
                    printf("> GPU does not support concurrent kernel execution\n");
                    printf(" CUDA kernel runs will be serialized\n");
                }
                break;
            }
        }
    }
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}

__global__
void fill(int n, bool* tab) {
    int index; index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride; stride = blockDim.x * gridDim.x;
    for (int i = index; i <= n; i += stride)
        tab[i] = true;
}

int main() {
    if (!InitCUDA())
    {
        return 0;
    }
    const int gpu_indices_for_count = 1024;
    int n;
    int* result;
    bool* tab;
    cout << "Podaj n: ";
    //cin >> n;
    n = 100000000;
    if (n > 3000000000)
    {
        cout << "input range too large, limit is 3*10e9";
        return 0;
    }
    auto start = high_resolution_clock::now();
    hipMallocManaged(&result, sizeof(int));
    hipMallocManaged(&tab, (n + 1) * sizeof(bool));
    fill<<<1000, gpu_indices_for_count >>>(n, tab);
    double sqrt_n = sqrt(n);
    hipDeviceSynchronize();
    fun << <1, 1024 >> > (n, sqrt_n, tab);
    hipDeviceSynchronize();
    sum << <1, gpu_indices_for_count >> > (n, tab, result);
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << endl << "Liczby pierwsze:" << endl;
    
    for (int i = 2; i <= n; i++)
        if (tab[i]) {
            //cout << i << ", ";
        }
    cout << endl << "Czas: " << duration.count() << endl;
    cout << "Liczb pierwszych w zakresie [2, n] jest: " << result[0];
    hipFree(result);
    hipFree(tab);
    //cin >> n;
    return 0;
}
